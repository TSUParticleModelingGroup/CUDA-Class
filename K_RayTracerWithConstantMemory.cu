#include "hip/hip_runtime.h"
// Name:
// Ray tracing
// nvcc 11RayTracerWithConstantMemory.cu -o temp -lglut -lGL -lm

/*
 What to do:
 This program creates a random set of spheres and uses ray tracing to render an image of them 
 to be displayed on the screen. In the scene, positive X is to the right, positive Y is up, and 
 positive Z comes out of the screen toward the viewer.

 All the spheres are located within a 2x2x2 cube, and you observe them through a 2x2 viewing window.
 
 Your mission, should you choose to accept it:
 1. The spheres created on the CPU do not change, so transfer them to the GPU and store them in constant memory.
 2. Use CUDA events to time your code execution.
*/

/*
 Purpose:
 To learn how to use constant memory and CUDA events.
*/

// Include files
#include <GL/glut.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>

// Defines
#define WINDOWWIDTH 1024
#define WINDOWHEIGHT 1024
#define XMIN -1.0f
#define XMAX 1.0f
#define YMIN -1.0f
#define YMAX 1.0f
#define ZMIN -1.0f
#define ZMAX 1.0f
#define NUMSPHERES 100
#define MAXRADIUS 0.2 // The biggest radius a sphere can have.

// Local structures
struct sphereStruct 
{
	float r,b,g; // Sphere color
	float radius;
	float x,y,z; // Sphere center
};

// Globals variables
static int Window;
unsigned int WindowWidth = WINDOWWIDTH;
unsigned int WindowHeight = WINDOWHEIGHT;
dim3 BlockSize, GridSize;
float *PixelsCPU, *PixelsGPU; 
sphereStruct *SpheresCPU, *SpheresGPU;

// Function prototypes
void cudaErrorCheck(const char *, int);
void Display();
void idle();
void KeyPressed(unsigned char , int , int );
__device__ float hit(float , float , float *, float , float , float , float );
__global__ void makeSphersBitMap(float *, sphereStruct *);
void makeRandomSpheres();
void makeBitMap();
void paintScreen();
void setup();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

void display()
{
	makeBitMap();	
}

void KeyPressed(unsigned char key, int x, int y)
{	
	if(key == 'q')
	{
		glutDestroyWindow(Window);
		
		// Free host memory.
		free(PixelsCPU); 
		free(SpheresCPU); 
	
		// Free divice memory.
		hipFree(PixelsGPU); 
		cudaErrorCheck(__FILE__, __LINE__);
		hipFree(SpheresGPU); 
		cudaErrorCheck(__FILE__, __LINE__);
		
		printf("\nw Good Bye\n");
		exit(0);
	}
}

__device__ float hit(float pixelx, float pixely, float *dimingValue, sphereStruct sphere)
{
	float dx = pixelx - sphere.x;  //Distance from ray to sphere center in x direction
	float dy = pixely - sphere.y;  //Distance from ray to sphere center in y direction
	float r2 = sphere.radius*sphere.radius;
	if(dx*dx + dy*dy < r2) // if the ray hits the sphere, then we need to find distance
	{
		float dz = sqrtf(r2 - dx*dx - dy*dy); // Distance from ray to edge of sphere?
		*dimingValue = dz/sphere.radius; // n is value between 0 and 1 used for darkening points near edge.
		return dz + sphere.z; //  Return the distance to be scaled by
	}
	return (ZMIN- 1.0); //If the ray doesn't hit anything return a number 1 unit behind the box.
}

__global__ void makeSphersBitMap(float *pixels, sphereStruct *sphereInfo)
{
	float stepSizeX = (XMAX - XMIN)/((float)WINDOWWIDTH - 1);
	float stepSizeY = (YMAX - YMIN)/((float)WINDOWHEIGHT - 1);
	
	// Asigning each thread a pixel
	float pixelx = XMIN + threadIdx.x*stepSizeX;
	float pixely = YMIN + blockIdx.x*stepSizeY;
	
	// Finding this pixels location in memory
	int id = 3*(threadIdx.x + blockIdx.x*blockDim.x);
	
	//initialize rgb values for each pixel to zero (black)
	float pixelr = 0.0f;
	float pixelg = 0.0f;
	float pixelb = 0.0f;
	float hitValue;
	float dimingValue;
	float maxHit = ZMIN -1.0f; // Initializing it to be 1 unit behind the box.
	for(int i = 0; i < NUMSPHERES; i++)
	{
		hitValue = hit(pixelx, pixely, &dimingValue, sphereInfo[i]);
		// do we hit any spheres? If so, how close are we to the center? (i.e. n)
		if(maxHit < hitValue)
		{
			// Setting the RGB value of the sphere but also diming it as it gets close to the side of the sphere.
			pixelr = sphereInfo[i].r * dimingValue; 	
			pixelg = sphereInfo[i].g * dimingValue;	
			pixelb = sphereInfo[i].b * dimingValue; 	
			maxHit = hitValue; // reset maxHit value to be the current closest sphere
		}
	}
	
	pixels[id] = pixelr;
	pixels[id+1] = pixelg;
	pixels[id+2] = pixelb;
}

void makeRandomSpheres()
{	
	float rangeX = XMAX - XMIN;
	float rangeY = YMAX - YMIN;
	float rangeZ = ZMAX - ZMIN;
	
	for(int i = 0; i < NUMSPHERES; i++)
	{
		SpheresCPU[i].x = (rangeX*(float)rand()/RAND_MAX) + XMIN;
		SpheresCPU[i].y = (rangeY*(float)rand()/RAND_MAX) + YMIN;
		SpheresCPU[i].z = (rangeZ*(float)rand()/RAND_MAX) + ZMIN;
		SpheresCPU[i].r = (float)rand()/RAND_MAX;
		SpheresCPU[i].g = (float)rand()/RAND_MAX;
		SpheresCPU[i].b = (float)rand()/RAND_MAX;
		SpheresCPU[i].radius = MAXRADIUS*(float)rand()/RAND_MAX;
	}
}	

void makeBitMap()
{	
	hipMemcpy(SpheresGPU, SpheresCPU, NUMSPHERES*sizeof(sphereStruct), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	makeSphersBitMap<<<GridSize, BlockSize>>>(PixelsGPU, SpheresGPU);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipMemcpyAsync(PixelsCPU, PixelsGPU, WINDOWWIDTH*WINDOWHEIGHT*3*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	
	paintScreen();
}

void paintScreen()
{
	//Putting pixels on the screen.
	glDrawPixels(WINDOWWIDTH, WINDOWHEIGHT, GL_RGB, GL_FLOAT, PixelsCPU); 
	glFlush();
}

void setup()
{
	//Allocating memory for the scene that will be displayed to the screen.
	//We need the 3 because each pixel has a red, green, and blue value.
	PixelsCPU = (float *)malloc(WINDOWWIDTH*WINDOWHEIGHT*3*sizeof(float));
	hipMalloc(&PixelsGPU,WINDOWWIDTH*WINDOWHEIGHT*3*sizeof(float)); 
	cudaErrorCheck(__FILE__, __LINE__);
	
	//Allocating memory for the spheres that will create the scene.
	//This is what you will be changing out for constant memory.
	SpheresCPU= (sphereStruct*)malloc(NUMSPHERES*sizeof(sphereStruct));
	hipMalloc(&SpheresGPU, NUMSPHERES*sizeof(sphereStruct));
	cudaErrorCheck(__FILE__, __LINE__);
	
	//Threads in a block
	if(WINDOWWIDTH > 1024) //To keep the code simple we make sure the scene width fits in a block.
	{
	 	printf("The window width is too large to run with this program\n");
	 	printf("The window width must be less than 1024.\n");
	 	printf("Good Bye and have a nice day!\n");
	 	exit(0);
	}
	BlockSize.x = WINDOWWIDTH;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	//Blocks in a grid
	GridSize.x = WINDOWHEIGHT;
	GridSize.y = 1;
	GridSize.z = 1;
	
	// Seeding the random number generator.
	time_t t;
	srand((unsigned) time(&t));
}

int main(int argc, char** argv)
{ 
	setup();
	makeRandomSpheres();
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(WINDOWWIDTH, WINDOWHEIGHT);
	Window = glutCreateWindow("Random Spheres");
	glutKeyboardFunc(KeyPressed);
   	glutDisplayFunc(display);
   	glutMainLoop();
}

